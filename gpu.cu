#include "hip/hip_runtime.h"
#include "gpu.hpp"
#include <thrust/sort.h>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void cu_grey(uint32_t *img, int w, int h) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

/*  int x = idx % w;
  int y = idx / w;

  if (x >= w)
    return;
  if (y >= h)
    return;*/

  if (idx > w * h)
    return;

  printf("idx: %d\n", idx);

  rgbx8888_u pix ={ .value = img[idx]};

  auto r = pix.argb.r;
  auto g = pix.argb.g;
  auto b = pix.argb.b;

  double grey_val = 0.3*(double)r + 0.59*(double)g + 0.11*(double)b;
  pix.argb.r = pix.argb.g = pix.argb.b = (unsigned char)((int)grey_val % 256);
  img[idx] = pix.value;
}

void gpu_grey(uint32_t *img, int w, int h) {
  size_t threadsByGrid = 1024;
  size_t gridNumber = ((w * h) / threadsByGrid) + 1;
  uint32_t *d_img = nullptr;

  std::cout
    << "threads: " << threadsByGrid << " "
    << "grids:   " << gridNumber << std::endl;
  hipMalloc(&d_img, w * h * sizeof(uint32_t));
  gpuErrchk(hipPeekAtLastError());
  hipMemcpy(d_img, img, w * h * sizeof(uint32_t), hipMemcpyHostToDevice);
  gpuErrchk(hipPeekAtLastError());

  cu_grey<<<gridNumber, threadsByGrid>>>(d_img, w, h);
  gpuErrchk(hipPeekAtLastError());

  hipMemcpy(img, d_img, w * h * sizeof(uint32_t), hipMemcpyDeviceToHost);
  gpuErrchk(hipPeekAtLastError());
  hipFree(d_img);
  gpuErrchk(hipPeekAtLastError());
}
